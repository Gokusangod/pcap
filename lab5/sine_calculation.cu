#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void calculateSine(float *angles, float *sines, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        sines[idx] = sin(angles[idx]);
    }
}

int main() {
    int n = 1024; // Length of the array
    size_t size = n * sizeof(float);

    // Allocate memory on the host
    float *h_angles = (float *)malloc(size);
    float *h_sines = (float *)malloc(size);

    // Initialize angles in radians
    for (int i = 0; i < n; i++) {
        h_angles[i] = static_cast<float>(i) * (M_PI / 180.0); // Convert degrees to radians
    }

    // Allocate memory on the device
    float *d_angles, *d_sines;
    hipMalloc(&d_angles, size);
    hipMalloc(&d_sines, size);

    // Copy angles from host to device
    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    calculateSine<<<blocksPerGrid, threadsPerBlock>>>(d_angles, d_sines, n);
    hipMemcpy(h_sines, d_sines, size, hipMemcpyDeviceToHost);

    // Print results for the first 10 sine values
    std::cout << "Sine values:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << h_sines[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_angles);
    hipFree(d_sines);

    // Free host memory
    free(h_angles);
    free(h_sines);

    return 0;
}
