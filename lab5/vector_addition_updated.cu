#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVectorsKernelNThreads(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 1024; // Length of the vectors
    size_t size = n * sizeof(float);

    // Allocate memory on the host
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = (float)i;
        h_b[i] = (float)i;
    }

    // Allocate memory on the device
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addVectorsKernelNThreads<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print results for n threads
    printf("Results for n threads:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_c[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
/*
$ nvcc vector_addition_updated.cu -o vector_addition_updated && ./vector_addition_updated
Results for n threads:
0.000000 2.000000 4.000000 6.000000 8.000000 10.000000 12.000000 14.000000 16.000000 18.000000 
*/
