#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addVectorsKernelNThreads(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 1024; // Length of the vectors
    size_t size = n * sizeof(float);

    // Allocate memory on the host
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i);
    }

    // Allocate memory on the device
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addVectorsKernelNThreads<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print results for n threads
    std::cout << "Results for n threads:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
